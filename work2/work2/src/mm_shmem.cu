#include "work2/mm_impls/kernel_mm_shmem.cuh"
#include "work2/mm_impls/mm_dispatch.hpp"
#include "work2/mm_impls/mm_shmem.hpp"

#include "hip/hip_runtime.h.hpp"

template <ScalarKind ScalarT>
void w2::matmul_shmem(const DeviceMatrix<ScalarT>& a,
    const DeviceMatrix<ScalarT>& b,
    DeviceMatrix<ScalarT>& c) {
  constexpr const dim3 block_size = {16, 16};

  const dim3 grid_size = {
      hip/hip_runtime.h::cover(b.size(1), block_size.x),
      hip/hip_runtime.h::cover(a.size(0), block_size.y),
  };

  kernel_mm_shmem<block_size.x><<<grid_size, block_size>>>(a.view(), b.view(), c.view());
}

MM_DISPATCH_FOR_ALL_SUPPORTED_TYPES(w2::matmul_shmem);
