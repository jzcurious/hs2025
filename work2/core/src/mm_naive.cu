#include "work2/mm_impls/dispatch_binary.hpp"
#include "work2/mm_impls/kernel_mm_naive.cuh"
#include "work2/mm_impls/mm_naive.hpp"

#include "hip/hip_runtime.h.hpp"

template <ScalarKind ScalarT>
MatrixView<ScalarT>& w2::matmul_naive(
    MatrixView<ScalarT>& c, const MatrixView<ScalarT>& a, const MatrixView<ScalarT>& b) {
  constexpr dim3 block_size = {16, 16};

  const dim3 grid_size = {
      hip/hip_runtime.h::cover(b.size(1), block_size.x),
      hip/hip_runtime.h::cover(a.size(0), block_size.y),
  };

  kernel_mm_naive<<<grid_size, block_size>>>(c, a, b);
  return c;
}

DISPATCH_BINARY_FOR_ALL_TYPES(w2::matmul_naive);
