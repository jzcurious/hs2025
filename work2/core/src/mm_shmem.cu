#include "work2/mm_impls/kernel_mm_shmem.cuh"
#include "work2/mm_impls/mm_dispatch.hpp"
#include "work2/mm_impls/mm_shmem.hpp"

#include "hip/hip_runtime.h.hpp"

template <ScalarKind ScalarT>
MatrixView<ScalarT>& w2::matmul_shmem(
    const MatrixView<ScalarT>& a, const MatrixView<ScalarT>& b, MatrixView<ScalarT>& c) {
  constexpr const dim3 block_size = {16, 16};

  const dim3 grid_size = {
      hip/hip_runtime.h::cover(b.size(1), block_size.x),
      hip/hip_runtime.h::cover(a.size(0), block_size.y),
  };

  kernel_mm_shmem<block_size.x><<<grid_size, block_size>>>(a, b, c);
  return c;
}

MM_DISPATCH_FOR_ALL_SUPPORTED_TYPES(w2::matmul_shmem);
