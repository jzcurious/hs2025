#include "work2/mm_impls/dispatch_binary.hpp"
#include "work4/add.hpp"
#include "work4/kernel_add.cuh"
#include "work4/kernel_broadcast_add.cuh"

#include "hip/hip_runtime.h.hpp"
#include <hip/hip_fp16.h>

template <ScalarKind ScalarT>
MatrixView<ScalarT>& w4::add(
    MatrixView<ScalarT>& c, const MatrixView<ScalarT>& a, const MatrixView<ScalarT>& b) {
  constexpr dim3 block_size = {16, 16};

  const dim3 grid_size = {
      hip/hip_runtime.h::cover(a.size(1), block_size.x),
      hip/hip_runtime.h::cover(a.size(0), block_size.y),
  };

  if (a.size(0) > b.size(0))
    kernel_broadcast_add<<<grid_size, block_size>>>(c, a, b);
  else
    kernel_add<<<grid_size, block_size>>>(c, a, b);

  return c;
}

DISPATCH_BINARY_FOR_ALL_TYPES(w4::add);
