#include "work3/mm_impls/kernel_mm_wmma.cuh"
#include "work4/dispatch_ternary.hpp"
#include "work4/graph_linear.hpp"
#include "work4/kernel_broadcast_add.cuh"

#include "hip/hip_runtime.h.hpp"

template <ScalarKind ScalarT>
MatrixView<ScalarT>& graph_linear(MatrixView<ScalarT>& y,
    const MatrixView<ScalarT>& x,
    const MatrixView<ScalarT>& w,
    const MatrixView<ScalarT>& b) {

  constexpr const dim3 block_size = {32, 1};
  constexpr const dim3 wmma_size = {16, 16, 16};

  const dim3 mm_grid_size = {
      hip/hip_runtime.h::cover(w.size(1), wmma_size.x),
      hip/hip_runtime.h::cover(x.size(0), wmma_size.y),
  };

  const dim3 add_grid_size = {
      hip/hip_runtime.h::cover(w.size(1), block_size.x),
      hip/hip_runtime.h::cover(x.size(0), block_size.y),
  };

  hipStream_t stream;
  hipStreamCreate(&stream);

  hipGraph_t graph;
  hipGraphExec_t graph_inst;

  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
  if (x.colmajor) {
    kernel_mm_wmma<MatrixView<ScalarT>, true, wmma_size.x, wmma_size.y, wmma_size.z>
        <<<mm_grid_size, block_size, 0, stream>>>(y, x, w);
  } else {
    kernel_mm_wmma<MatrixView<ScalarT>, false, wmma_size.x, wmma_size.y, wmma_size.z>
        <<<mm_grid_size, block_size, 0, stream>>>(y, x, w);
  }
  kernel_broadcast_add<<<add_grid_size, block_size, 0, stream>>>(y, y, b);
  hipStreamEndCapture(stream, &graph);

  hipGraphInstantiate(&graph_inst, graph);

  hipGraphLaunch(graph_inst, stream);
  hipStreamSynchronize(stream);

  hipGraphExecDestroy(graph_inst);
  hipGraphDestroy(graph);
  hipStreamDestroy(stream);

  return y;
}

DISPATCH_TERNARY(graph_linear, half);

#if __CUDA_ARCH__ >= 800
DISPATCH_TERNARY(graph_linear, float);
#endif
